#include "hip/hip_runtime.h"
#include<stdio.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#define N 32
#define M 8 //Watki na blok

__global__ void add(int *a, int *b, int *c) 
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
        printf("INDEX: %d, THREAD: %d, BLOCK: %d\n", index, threadIdx.x, blockIdx.x);
	c[index] = a[index] + b[index];
}

void random (int *tab, int wym )
{	
	int i;
	for(i=0;i<wym;i++)
		tab[i]=rand()%101;
}


int main(void) {
	int *a, *b, *c; // host copies of a, b, c
	int *d_a, *d_b, *d_c; // device copies of a, b, c
	int size = N * sizeof(int);
	int i;
	srand(time(NULL));
	// Allocate space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	// Alloc space for host copies of a, b, c and setup input values
	a = (int *)malloc(size); random(a, N);
	b = (int *)malloc(size); random(b, N);
	c = (int *)malloc(size);
	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	// Launch add() kernel on GPU
	add<<<N/M,M>>>(d_a, d_b, d_c);
	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	for(i=0;i<N;i++)
	{
		printf("a[%d](%d) + b[%d](%d) = c[%d](%d)\n",i,a[i],i,b[i],i,c[i]);
	}
	// Cleanup
	//printf("%d+%d=%d\n",a,b,c);
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}


